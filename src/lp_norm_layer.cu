#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/lp_norm_layer.hpp"

namespace caffe {

template <typename Dtype>
void L2NormLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const int num = bottom[0]->shape(0);
  const int dim = count / num;
	const Dtype eps = 1e-9;
  Dtype * inv_norm = this->inv_lpnorm_;

  for (int i=0; i<num; ++i) {
    const Dtype * bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(i);
    caffe_gpu_dot(dim, bottom_data, bottom_data, inv_norm + i);
  }
  for (int i=0; i<num; ++i) {
    const Dtype * bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(i);
    Dtype * top_data = top[0]->mutable_gpu_data() + top[0]->offset(i);
    inv_norm[i] = 1.0 / (std::sqrt(inv_norm[i])+eps);
    caffe_gpu_scale(dim, inv_norm[i], bottom_data, top_data);
  }
}

template <typename Dtype>
void L2NormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  const int count = bottom[0]->count();
  const int num = bottom[0]->shape(0);
  const int dim = count / num;

  Dtype * inv_norm = this->inv_lpnorm_;
  for (int i=0; i<num; ++i) {
    const Dtype* top_data = top[0]->gpu_data() + top[0]->offset(i);
    Dtype* top_diff = top[0]->mutable_gpu_diff() + top[0]->offset(i);
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff() + bottom[0]->offset(i);

    Dtype temp;
    caffe_gpu_scale(dim, inv_norm[i], top[0]->gpu_diff() + top[0]->offset(i), top_diff);
    caffe_gpu_dot(dim, top_data, top[0]->gpu_diff() + top[0]->offset(i), &temp);
    caffe_gpu_scale(dim, -temp, top_data, bottom_diff);
    caffe_gpu_add(dim, bottom[0]->gpu_diff() + bottom[0]->offset(i), top[0]->gpu_diff() + top[0]->offset(i), bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormLayer);


}  // namespace caffe
