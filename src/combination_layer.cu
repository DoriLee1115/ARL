#include <vector>

#include "caffe/layers/combination_layer.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {


/* copy only clipped region */
template <typename Dtype>
void CombinationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

	Forward_cpu(bottom,top);
  
}
/* copy only clipped region */

template <typename Dtype>
void CombinationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(CombinationLayer);

}

